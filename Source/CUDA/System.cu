#include "hip/hip_runtime.h"
#include <cutil.h>
#include <GL/glut.h>
#include <cuda_gl_interop.h>

#include "Kernel.cu"
#include "radixsort.cu"


extern "C"
{

//  Utility, data
#define CU  CUDA_SAFE_CALL

#define cuMapVbo(pv,vbo)		CU(cudaGLMapBufferObject((void**)&pv, vbo));
#define cuUnMapVbo(vbo)			CU(cudaGLUnmapBufferObject(vbo));

#define cuBindTex(tex,pv,size)	CU(hipBindTexture(0, tex, pv, size));
#define cuUnbindTex(tex)		CU(hipUnbindTexture(tex));


void cudaInit(int argc, char **argv)	{   CUT_DEVICE_INIT(argc, argv);	}
bool cudaInitB(int argc, char **argv, bool showInfo)	//  no exit + info
{
	int deviceCount;
	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0)  {
		fprintf(stderr, "Error: no devices supporting CUDA.\n");
		return false;  }  //exit(EXIT_FAILURE);

	int dev = 0;
	cutGetCmdLineArgumenti(argc, (const char**)argv, "device", &dev);
	if (dev > deviceCount-1) dev = deviceCount - 1;
	hipDeviceProp_t p;

	CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&p, dev));
	if (p.major < 1)  {
		fprintf(stderr, "Error: device does not support CUDA.\n");
		return false;	}
    if (dev == 0)  {
        if (p.major == 9999 && p.minor == 9999)
            printf("There is no device supporting CUDA.\n");
        else  if (deviceCount != 1)
            printf("There are %d devices supporting CUDA\n", deviceCount);  }
	
	if (cutCheckCmdLineFlag(argc, (const char**)argv, "quiet") == CUTFalse)
		fprintf(stderr, "Using device %d: %s\n\n", dev, p.name);
		
	if (showInfo)
	{
        printf("CUDA version:  %d.%d\n", p.major, p.minor);
        printf("Total memory:  %u bytes  (%u MB)\n", p.totalGlobalMem, p.totalGlobalMem/1024/1024);
		#if CUDART_VERSION >= 2000
        printf("Multiprocessors:  %d  Cores:  %d\n", p.multiProcessorCount, 8*p.multiProcessorCount);
		#endif
		printf("Clock rate:  %.4f GHz\n", p.clockRate * 1e-6f);
		printf(" Warp size:  %d\n", p.warpSize);
		printf(" Constant memory:  %u bytes\n", p.totalConstMem); 
		printf("  Shared memory per block:  %u bytes\n", p.sharedMemPerBlock);
		printf("  Registers per block:   %d\n", p.regsPerBlock);
		printf("  Max threads per block: %d\n", p.maxThreadsPerBlock);
		printf("  Max sizes of a block:  %d x %d x %d\n", p.maxThreadsDim[0], p.maxThreadsDim[1], p.maxThreadsDim[2]);
		printf("  Max sizes of a grid:   %d x %d x %d\n", p.maxGridSize[0], p.maxGridSize[1], p.maxGridSize[2]);
		printf("  Max memory pitch:    %u bytes\n", p.memPitch);
		printf("  Texture alignment:   %u bytes\n", p.textureAlignment);
	    #if CUDART_VERSION >= 2000
        printf("  Concurrent copy and execution:  %s\n\n", p.deviceOverlap ? "Yes" : "No");
		#endif
	}
	
	CUDA_SAFE_CALL(hipSetDevice(dev));
	return true;
}
void threadSync()						{	CU(hipDeviceSynchronize());	}


void allocateArray(void **devPtr, size_t size)	{	CU(hipMalloc(devPtr, size));	}
void freeArray(void *devPtr)					{	CU(hipFree(devPtr));	}

void registerGLvbo(uint vbo)		{	CU(cudaGLRegisterBufferObject(vbo));	}
void unregGLvbo(uint vbo)			{	CU(cudaGLUnregisterBufferObject(vbo));	}


void copyFromDevice(void* host, const void* device, unsigned int vbo, int size)
{
	if (vbo)  cuMapVbo(device, vbo);
	CU(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
	if (vbo)  cuUnMapVbo(vbo);
}
void copyToDevice(void* device, const void* host, int offset, int size)
{
	CU(hipMemcpy((char*) device + offset, host, size, hipMemcpyHostToDevice));
}

void setParameters(SimParams *hostParams)
{	// copy parameters to constant memory
	CU(hipMemcpyToSymbol(HIP_SYMBOL(par), hostParams, sizeof(SimParams)));
}


//  Round a / b to nearest higher integer value
int iDivUp(int a, int b) {	return a%b != 0 ? a/b+1 : a/b;	}

//  compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
	numThreads = min(blockSize, n);  numBlocks = iDivUp(n, numThreads);
}
//----------------------------------------------------------------------------------



///  integrate
void integrate(uint vboOldPos, uint vboNewPos, float4* oldVel, float4* newVel, int numParticles)
{
	int numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);

	float4 *oldPos, *newPos;
	cuMapVbo(oldPos, vboOldPos);  cuMapVbo(newPos, vboNewPos);

	integrateD<<< numBlocks, numThreads >>>(newPos, newVel, oldPos, oldVel);

	CUT_CHECK_ERROR("Kernel execution failed: Integrate");
	cuUnMapVbo(vboOldPos);  cuUnMapVbo(vboNewPos);

	/***/threadSync();
}


///  calcHash
void calcHash(uint vboPos, uint2* particleHash, int numParticles)
{
	int numThreads, numBlocks;
	computeGridSize(numParticles, 2*256, numBlocks, numThreads);

	float4 *pos;
	cuMapVbo(pos, vboPos);

	calcHashD<<< numBlocks, numThreads >>>(pos, particleHash);

	CUT_CHECK_ERROR("Kernel execution failed: calcHash");
	cuUnMapVbo(vboPos);

	/***/threadSync();
}


///  reorder
void reorder(uint vboOldPos, float4* oldVel, float4* sortedPos, float4* sortedVel,
		uint2* particleHash, uint* cellStart,  uint numParticles, uint numCells)
{
	int numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);
	CU(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

	float4 *oldPos;
	cuMapVbo(oldPos, vboOldPos);
	#if USE_TEX
	uint spar4 = numParticles*sizeof(float4);
	cuBindTex(oldPosTex, oldPos, spar4);	cuBindTex(oldVelTex, oldVel, spar4);
	#endif

	reorderD<<< numBlocks, numThreads >>>(particleHash, cellStart,
		oldPos, oldVel, sortedPos, sortedVel);

	CUT_CHECK_ERROR("Kernel execution failed: reorder");
	#if USE_TEX
	cuUnbindTex(oldPosTex);  cuUnbindTex(oldVelTex);
	#endif
	cuUnMapVbo(vboOldPos);

	/***/threadSync();
}


///  collide
void collide(uint timer,  uint vboOldPos, uint vboNewPos, /**/uint vboCLR2,
		float4* sortedPos, float4* sortedVel, float4* oldVel, float4* newVel,
		float* pressure, float* density, float* dyeColor,
		uint2* particleHash, uint* cellStart, uint numParticles, uint numCells)
{
	/**/CU(cutStartTimer(timer));
	float4 *oldPos, *newPos, *clr2;
	cuMapVbo(clr2, vboCLR2);
	cuMapVbo(oldPos, vboOldPos);  cuMapVbo(newPos, vboNewPos);  

	#if USE_TEX
	uint spar4 = numParticles*sizeof(float4), spar = numParticles*sizeof(float);
	cuBindTex(oldPosTex, sortedPos, spar4);  cuBindTex(pressureTex, pressure, spar);
	cuBindTex(oldVelTex, sortedVel, spar4);	 cuBindTex(densityTex, density, spar);  cuBindTex(dyeColorTex, dyeColor, spar);//
	cuBindTex(particleHashTex, particleHash, numParticles*sizeof(uint2));
	cuBindTex(cellStartTex, cellStart, numCells*sizeof(uint));
	#endif


	int numThreads, numBlocks;
	computeGridSize(numParticles, 64, numBlocks, numThreads);
	

	computeDensityD<<< numBlocks, numThreads >>>(clr2, sortedPos,  pressure, density,  particleHash, cellStart);
	/***/threadSync();
	/**/CU(cutStopTimer(timer));

	  computeForceD<<< numBlocks, numThreads >>>(newPos, newVel, sortedPos, sortedVel,  clr2, pressure, density, dyeColor/**/, particleHash, cellStart);
		//collideD<<< numBlocks, numThreads >>>(newPos, newVel, sortedPos, sortedVel, /**/clr2,  particleHash, cellStart);

	CUT_CHECK_ERROR("Kernel execution failed: Collide");

	cuUnMapVbo(vboNewPos);  cuUnMapVbo(vboOldPos);  cuUnMapVbo(vboCLR2);
	#if USE_TEX
	cuUnbindTex(oldPosTex);		cuUnbindTex(oldVelTex);
	cuUnbindTex(pressureTex)	cuUnbindTex(densityTex);	cuUnbindTex(dyeColorTex);//
	cuUnbindTex(particleHashTex);	cuUnbindTex(cellStartTex);
	#endif
	/***/threadSync();
}


}  //extern "C"